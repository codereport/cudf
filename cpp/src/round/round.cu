#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_factories.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/round.hpp>
#include <cudf/round.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <type_traits>

namespace cudf {
namespace detail {
namespace {  // anonymous

float __device__ generic_round(float f) { return roundf(f); }
double __device__ generic_round(double d) { return ::round(d); }

float __device__ generic_round_half_even(float f) { return rintf(f); }
double __device__ generic_round_half_even(double d) { return ::rint(d); }

float __device__ generic_modf(float a, float* b) { return modff(a, b); }
double __device__ generic_modf(double a, double* b) { return modf(a, b); }

template <typename T, typename std::enable_if_t<std::is_signed<T>::value>* = nullptr>
T __device__ generic_abs(T value)
{
  return abs(value);
}

template <typename T, typename std::enable_if_t<not std::is_signed<T>::value>* = nullptr>
T __device__ generic_abs(T value)
{
  return value;
}

template <typename T, typename std::enable_if_t<std::is_signed<T>::value>* = nullptr>
bool __device__ is_negative(T value)
{
  return value < 0;
}

// this is needed to suppress warning: pointless comparison of unsigned integer with zero
template <typename T, typename std::enable_if_t<not std::is_signed<T>::value>* = nullptr>
bool __device__ is_negative(T)
{
  return false;
}

template <typename T>
struct HalfUpZero {
  T n;  // unused in the decimal_places = 0 case
  template <typename U = T, typename std::enable_if_t<cudf::is_floating_point<U>()>* = nullptr>
  __device__ U operator()(U e)
  {
    return generic_round(e);
  }

  template <typename U = T, typename std::enable_if_t<std::is_integral<U>::value>* = nullptr>
  __device__ U operator()(U e)
  {
    assert(false);  // Should never get here. Just for compilation
    return U{};
  }
};

template <typename T>
struct HalfUpPositive {
  T n;
  template <typename U = T, typename std::enable_if_t<cudf::is_floating_point<U>()>* = nullptr>
  __device__ U operator()(U e)
  {
    T integer_part;
    T const fractional_part = generic_modf(e, &integer_part);
    return integer_part + generic_round(fractional_part * n) / n;
  }

  template <typename U = T, typename std::enable_if_t<std::is_integral<U>::value>* = nullptr>
  __device__ U operator()(U e)
  {
    assert(false);  // Should never get here. Just for compilation
    return U{};
  }
};

template <typename T>
struct HalfUpNegative {
  T n;
  template <typename U = T, typename std::enable_if_t<cudf::is_floating_point<U>()>* = nullptr>
  __device__ U operator()(U e)
  {
    return generic_round(e / n) * n;
  }

  template <typename U = T, typename std::enable_if_t<std::is_integral<U>::value>* = nullptr>
  __device__ U operator()(U e)
  {
    auto const down = (e / n) * n;  // result from rounding down
    auto const sign = is_negative(e) ? -1 : 1;
    return down + sign * (generic_abs(e - down) >= n / 2 ? n : 0);
  }
};

template <typename T>
struct HalfEvenZero {
  T n;  // unused in the decimal_places = 0 case
  template <typename U = T, typename std::enable_if_t<cudf::is_floating_point<U>()>* = nullptr>
  __device__ U operator()(U e)
  {
    return generic_round_half_even(e);
  }

  template <typename U = T, typename std::enable_if_t<std::is_integral<U>::value>* = nullptr>
  __device__ U operator()(U e)
  {
    assert(false);  // Should never get here. Just for compilation
    return U{};
  }
};

template <typename T>
struct HalfEvenPositive {
  T n;
  template <typename U = T, typename std::enable_if_t<cudf::is_floating_point<U>()>* = nullptr>
  __device__ U operator()(U e)
  {
    T integer_part;
    T const fractional_part = generic_modf(e, &integer_part);
    return integer_part + generic_round_half_even(fractional_part * n) / n;
  }

  template <typename U = T, typename std::enable_if_t<std::is_integral<U>::value>* = nullptr>
  __device__ U operator()(U e)
  {
    assert(false);  // Should never get here. Just for compilation
    return U{};
  }
};

template <typename T>
struct HalfEvenNegative {
  T n;
  template <typename U = T, typename std::enable_if_t<cudf::is_floating_point<U>()>* = nullptr>
  __device__ U operator()(U e)
  {
    return generic_round_half_even(e / n) * n;
  }

  template <typename U = T, typename std::enable_if_t<std::is_integral<U>::value>* = nullptr>
  __device__ U operator()(U e)
  {
    assert(false);  // TODO support
    return U{};
  }
};

template <typename T, typename RoundFunctor>
std::unique_ptr<column> round_with(column_view const& input,
                                   int32_t decimal_places,
                                   hipStream_t stream,
                                   rmm::mr::device_memory_resource* mr)
{
  if (decimal_places >= 0 && std::is_integral<T>::value)
    return std::make_unique<cudf::column>(input, stream, mr);

  auto result = cudf::make_fixed_width_column(input.type(),  //
                                              input.size(),
                                              copy_bitmask(input, stream, mr),
                                              input.null_count(),
                                              stream,
                                              mr);

  auto out_view = result->mutable_view();
  T const n     = std::pow(10, std::abs(decimal_places));

  thrust::transform(rmm::exec_policy(stream)->on(stream),
                    input.begin<T>(),
                    input.end<T>(),
                    out_view.begin<T>(),
                    RoundFunctor{n});

  return result;
}

struct round_type_dispatcher {
  template <typename T, typename... Args>
  std::enable_if_t<not cudf::is_numeric<T>(), std::unique_ptr<column>> operator()(Args&&... args)
  {
    CUDF_FAIL("Type not support for cudf::round");
  }

  template <typename T>
  std::enable_if_t<cudf::is_numeric<T>(), std::unique_ptr<column>> operator()(
    column_view const& input,
    int32_t decimal_places,
    cudf::rounding_method method,
    hipStream_t stream,
    rmm::mr::device_memory_resource* mr)
  {
    // clang-format off
    switch (method) {
      case cudf::rounding_method::HALF_UP:
        if      (decimal_places == 0) return round_with<T, HalfUpZero    <T>>(input, decimal_places, stream, mr);
        else if (decimal_places  > 0) return round_with<T, HalfUpPositive<T>>(input, decimal_places, stream, mr);
        else                          return round_with<T, HalfUpNegative<T>>(input, decimal_places, stream, mr);
      case cudf::rounding_method::HALF_EVEN:
        if      (decimal_places == 0) return round_with<T, HalfEvenZero    <T>>(input, decimal_places, stream, mr);
        else if (decimal_places >  0) return round_with<T, HalfEvenPositive<T>>(input, decimal_places, stream, mr);
        else                          return round_with<T, HalfEvenNegative<T>>(input, decimal_places, stream, mr);
      default: CUDF_FAIL("Undefined rounding method");
    }
    // clang-format on
  }
};

};  // anonymous namespace

std::unique_ptr<column> round(column_view const& input,
                              int32_t decimal_places,
                              cudf::rounding_method method,
                              hipStream_t stream,
                              rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(cudf::is_numeric(input.type()), "Only integral/floating point currently supported.");

  // TODO when fixed_point supported, have to adjust type
  if (input.size() == 0) return empty_like(input);

  return type_dispatcher(
    input.type(), round_type_dispatcher{}, input, decimal_places, method, stream, mr);
}

}  // namespace detail

std::unique_ptr<column> round(column_view const& input,
                              int32_t decimal_places,
                              rounding_method method,
                              rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return cudf::detail::round(input, decimal_places, method, 0, mr);
}

}  // namespace cudf
